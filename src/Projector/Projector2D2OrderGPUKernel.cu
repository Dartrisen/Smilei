
#include <hip/hip_runtime.h>
#if defined( SMILEI_ACCELERATOR_GPU_OMP )

    //! Simple switch to jump between the reference (omp) implementation and the
    //! hip one.
    //! NOTE: If you wanna use the OMP version, you must rename this file to
    //! .cpp instead of .cu for the HIP. The preprocessor and the Smilei
    //! makefile will take care of the rest.
    //!
    #if defined( __HIP__ )
    // HIP compiler support enabled (for .cu files)
    #else
        #define PRIVATE_SMILEI_USE_OPENMP_PROJECTION_IMPLENTATION = 1
    #endif

    #if defined( PRIVATE_SMILEI_USE_OPENMP_PROJECTION_IMPLENTATION )
        #include <cmath>

        #include "Tools.h"
    #else
        #include <hip/hip_runtime.h>

        #include "Params.h"
        #include "gpu.h"
    #endif

    #if defined( PRIVATE_SMILEI_USE_OPENMP_PROJECTION_IMPLENTATION )

namespace naive {

    static inline void
    currentDepositionKernel( double *__restrict__ Jx,
                             double *__restrict__ Jy,
                             double *__restrict__ Jz,
                             int Jx_size,
                             int Jy_size,
                             int Jz_size,
                             const double *__restrict__ device_particle_position_x,
                             const double *__restrict__ device_particle_position_y,
                             const double *__restrict__ device_particle_momentum_z,
                             const short *__restrict__ device_particle_charge,
                             const double *__restrict__ device_particle_weight,
                             const int *__restrict__ host_bin_index,
                             unsigned int x_dimension_bin_count,
                             unsigned int y_dimension_bin_count,
                             const double *__restrict__ invgf_,
                             const int *__restrict__ iold_,
                             const double *__restrict__ deltaold_,
                             double inv_cell_volume,
                             double dx_inv,
                             double dy_inv,
                             double dx_ov_dt,
                             double dy_ov_dt,
                             int    i_domain_begin,
                             int    j_domain_begin,
                             int    nprimy,
                             int    pxr )
    {
        const unsigned int bin_count = x_dimension_bin_count * y_dimension_bin_count;

        SMILEI_ASSERT( bin_count > 0 );

        const int particle_count = host_bin_index[bin_count - 1];

            // // Arrays used for the Esirkepov projection method
            // static constexpr bool kAutoDeviceFree = true;
            // const std::size_t     kTmpArraySize   = particle_count * 5;

            // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> Sx0_buffer{ kTmpArraySize };
            // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> Sx1_buffer{ kTmpArraySize };
            // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> Sy0_buffer{ kTmpArraySize };
            // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> Sy1_buffer{ kTmpArraySize };
            // // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> DSx_buffer{ kTmpArraySize };
            // // smilei::tools::gpu::NonInitializingVector<double, kAutoDeviceFree> DSy_buffer{ kTmpArraySize };

            // double *const __restrict__ Sx0_buffer_data = Sx0_buffer.data();
            // double *const __restrict__ Sx1_buffer_data = Sx1_buffer.data();
            // double *const __restrict__ Sy0_buffer_data = Sy0_buffer.data();
            // double *const __restrict__ Sy1_buffer_data = Sy1_buffer.data();
            // // double *const __restrict__ DSx_buffer_data = DSx_buffer.data();
            // // double *const __restrict__ DSy_buffer_data = DSy_buffer.data();

        #pragma omp target     is_device_ptr /* map */ ( /* to: */                                            \
                                                     device_particle_position_x /* [0:particle_count] */, \
                                                     device_particle_position_y /* [0:particle_count] */, \
                                                     device_particle_momentum_z /* [0:particle_count] */, \
                                                     device_particle_charge /* [0:particle_count] */,     \
                                                     device_particle_weight /* [0:particle_count] */ )
        #pragma omp teams thread_limit( 64 )
        #pragma omp distribute parallel for
        for( int particle_index = 0; particle_index < particle_count; ++particle_index ) {
            const double invgf                        = invgf_[particle_index];
            const int *const __restrict__ iold        = &iold_[particle_index];
            const double *const __restrict__ deltaold = &deltaold_[particle_index];

            double Sx0[5];
            double Sx1[5];
            double Sy0[5];
            double Sy1[5];
            // double DSx[5];
            // double DSy[5];

            // double *const __restrict__ Sx0 = Sx0_buffer_data + 5 * ( particle_index - 0 );
            // double *const __restrict__ Sx1 = Sx1_buffer_data + 5 * ( particle_index - 0 );
            // double *const __restrict__ Sy0 = Sy0_buffer_data + 5 * ( particle_index - 0 );
            // double *const __restrict__ Sy1 = Sy1_buffer_data + 5 * ( particle_index - 0 );
            // // double *const __restrict__ DSx = DSx_buffer_data + 5 * ( particle_index - 0 );
            // // double *const __restrict__ DSy = DSy_buffer_data + 5 * ( particle_index - 0 );

            // Variable declaration & initialization
            // Esirkepov's paper: https://arxiv.org/pdf/physics/9901047.pdf

            // Locate the particle on the primal grid at former time-step & calculate coeff. S0
            {
                const double delta  = deltaold[0 * particle_count];
                const double delta2 = delta * delta;
                Sx0[0]              = 0.0;
                Sx0[1]              = 0.5 * ( delta2 - delta + 0.25 );
                Sx0[2]              = 0.75 - delta2;
                Sx0[3]              = 0.5 * ( delta2 + delta + 0.25 );
                Sx0[4]              = 0.0;
            }
            {
                const double delta  = deltaold[1 * particle_count];
                const double delta2 = delta * delta;
                Sy0[0]              = 0.0;
                Sy0[1]              = 0.5 * ( delta2 - delta + 0.25 );
                Sy0[2]              = 0.75 - delta2;
                Sy0[3]              = 0.5 * ( delta2 + delta + 0.25 );
                Sy0[4]              = 0.0;
            }

            // Locate the particle on the primal grid at current time-step & calculate coeff. S1
            {
                const double xpn      = device_particle_position_x[particle_index] * dx_inv;
                const int    ip       = std::round( xpn );
                const int    ipo      = iold[0 * particle_count];
                const int    ip_m_ipo = ip - ipo - i_domain_begin;
                const double delta    = xpn - static_cast<double>( ip );
                const double delta2   = delta * delta;

                Sx1[0] = 0.0;
                Sx1[1] = 0.0;
                // Sx1[2] = 0.0; // Always set below
                Sx1[3] = 0.0;
                Sx1[4] = 0.0;

                Sx1[ip_m_ipo + 1] = 0.5 * ( delta2 - delta + 0.25 );
                Sx1[ip_m_ipo + 2] = 0.75 - delta2;
                Sx1[ip_m_ipo + 3] = 0.5 * ( delta2 + delta + 0.25 );
            }
            {
                const double ypn      = device_particle_position_y[particle_index] * dy_inv;
                const int    jp       = std::round( ypn );
                const int    jpo      = iold[1 * particle_count];
                const int    jp_m_jpo = jp - jpo - j_domain_begin;
                const double delta    = ypn - static_cast<double>( jp );
                const double delta2   = delta * delta;

                Sy1[0] = 0.0;
                Sy1[1] = 0.0;
                // Sy1[2] = 0.0; // Always set below
                Sy1[3] = 0.0;
                Sy1[4] = 0.0;

                Sy1[jp_m_jpo + 1] = 0.5 * ( delta2 - delta + 0.25 );
                Sy1[jp_m_jpo + 2] = 0.75 - delta2;
                Sy1[jp_m_jpo + 3] = 0.5 * ( delta2 + delta + 0.25 );
            }

            // DSx[0] = Sx1[0] - Sx0[0];
            // DSx[1] = Sx1[1] - Sx0[1];
            // DSx[2] = Sx1[2] - Sx0[2];
            // DSx[3] = Sx1[3] - Sx0[3];
            // DSx[4] = Sx1[4] - Sx0[4];

            // DSy[0] = Sy1[0] - Sy0[0];
            // DSy[1] = Sy1[1] - Sy0[1];
            // DSy[2] = Sy1[2] - Sy0[2];
            // DSy[3] = Sy1[3] - Sy0[3];
            // DSy[4] = Sy1[4] - Sy0[4];
            // }

            // // Charge deposition on the grid

            // for( int particle_index = 0; particle_index < particle_count; ++particle_index ) {
            //     const double invgf                        = invgf_[particle_index];
            //     const int *const __restrict__ iold        = &iold_[particle_index];
            //     const double *const __restrict__ deltaold = &deltaold_[particle_index];

            //     double *const __restrict__ Sx0 = Sx0_buffer_data + 5 * ( particle_index - 0 );
            //     double *const __restrict__ Sx1 = Sx1_buffer_data + 5 * ( particle_index - 0 );
            //     double *const __restrict__ Sy0 = Sy0_buffer_data + 5 * ( particle_index - 0 );
            //     double *const __restrict__ Sy1 = Sy1_buffer_data + 5 * ( particle_index - 0 );
            //     // double *const __restrict__ DSx = DSx_buffer_data + 5 * ( particle_index - 0 );
            //     // double *const __restrict__ DSy = DSy_buffer_data + 5 * ( particle_index - 0 );

            // (x,y,z) components of the current density for the macro-particle
            const double charge_weight = inv_cell_volume * static_cast<double>( device_particle_charge[particle_index] ) * device_particle_weight[particle_index];
            const double crx_p         = charge_weight * dx_ov_dt;
            const double cry_p         = charge_weight * dy_ov_dt;
            const double crz_p         = charge_weight * ( 1.0 / 3.0 ) * device_particle_momentum_z[particle_index] * invgf;

            // This is the particle position as grid index
            // This minus 2 come from the order 2 scheme, based on a 5 points stencil from -2 to +2.
            const int ipo = iold[0 * particle_count] - 2;
            const int jpo = iold[1 * particle_count] - 2;

            for( unsigned int i = 0; i < 1; ++i ) {
                const int iloc = ( i + ipo ) * nprimy + jpo;
                    /* Jx[iloc] += tmpJx[0]; */
        #pragma omp atomic update
                Jz[iloc] += crz_p * ( Sy1[0] * ( /* 0.5 * Sx0[i] + */ Sx1[i] ) );
                double tmp = 0.0;
                for( unsigned int j = 1; j < 5; j++ ) {
                    tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + 0.5 * ( Sx1[i] - Sx0[i] ) );
        #pragma omp atomic update
                    Jy[iloc + j + pxr * ( /* i + */ ipo )] += tmp;
        #pragma omp atomic update
                    Jz[iloc + j] += crz_p * ( Sy0[j] * ( 0.5 * Sx1[i] /* + Sx0[i] */ ) +
                                              Sy1[j] * ( /* 0.5 * Sx0[i] + */ Sx1[i] ) );
                }
            }

            double tmpJx[5]{};

            for( unsigned int i = 1; i < 5; ++i ) {
                const int iloc = ( i + ipo ) * nprimy + jpo;
                tmpJx[0] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( 0.5 * ( Sy1[0] - Sy0[0] ) );
        #pragma omp atomic update
                Jx[iloc] += tmpJx[0];
        #pragma omp atomic update
                Jz[iloc] += crz_p * ( Sy1[0] * ( 0.5 * Sx0[i] + Sx1[i] ) );
                double tmp = 0.0;
                for( unsigned int j = 1; j < 5; ++j ) {
                    tmpJx[j] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( Sy0[j] + 0.5 * ( Sy1[j] - Sy0[j] ) );
        #pragma omp atomic update
                    Jx[iloc + j] += tmpJx[j];
                    tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + 0.5 * ( Sx1[i] - Sx0[i] ) );
        #pragma omp atomic update
                    Jy[iloc + j + pxr * ( i + ipo )] += tmp;

        #pragma omp atomic update
                    Jz[iloc + j] += crz_p * ( Sy0[j] * ( 0.5 * Sx1[i] + Sx0[i] ) +
                                              Sy1[j] * ( 0.5 * Sx0[i] + Sx1[i] ) );
                }
            }
        }
    }
} // namespace naive

    #else

namespace hip {
    namespace detail {
        void checkErrors( ::hipError_t an_error_code,
                          const char  *file_name,
                          int          line )
        {
            if( an_error_code != ::hipError_t::hipSuccess ) {
                std::cout << "HIP error at " << file_name << ":" << line
                          << " -> " << ::hipGetErrorString( an_error_code );
                std::exit( EXIT_FAILURE );
            }
        }
    } // namespace detail

        #define checkHIPErrors( an_expression )                           \
            do {                                                          \
                detail::checkErrors( an_expression, __FILE__, __LINE__ ); \
            } while( 0 )

    namespace kernel {

        template <typename T>
        __device__ void
        atomicAdd( T *a_pointer, T a_value )
        {
            ::atomicAdd( a_pointer, a_value ); // Atomic     | the kernel is correct
            *a_pointer = a_value;              // Not atomic | the kernel is wrong but 10x faster
        }

        template <typename Float>
        __global__ void
        // __launch_bounds__(128, 4)
        depositeForAllCurrentDimensions( double *__restrict__ device_Jx,
                                         double *__restrict__ device_Jy,
                                         double *__restrict__ device_Jz,
                                         int Jx_size,
                                         int Jy_size,
                                         int Jz_size,
                                         const double *__restrict__ device_particle_position_x,
                                         const double *__restrict__ device_particle_position_y,
                                         const double *__restrict__ device_particle_momentum_z,
                                         const short *__restrict__ device_particle_charge,
                                         const double *__restrict__ device_particle_weight,
                                         const int *__restrict__ device_bin_index,
                                         const double *__restrict__ device_invgf_,
                                         const int *__restrict__ device_iold_,
                                         const double *__restrict__ device_deltaold_,
                                         Float inv_cell_volume,
                                         Float dx_inv,
                                         Float dy_inv,
                                         Float dx_ov_dt,
                                         Float dy_ov_dt,
                                         int   i_domain_begin,
                                         int   j_domain_begin,
                                         int   nprimy,
                                         int   pxr )
        {
            // TODO(Etienne M): refactor this function. Break it into smaller
            // pieces (lds init/store, coeff computation, deposition etc..)
            // TODO(Etienne M): prefer unsigned int vs int. At least the reader
            // knows the value wont be negative.
            const unsigned int workgroup_size = blockDim.x;
            const unsigned int bin_count      = gridDim.x * gridDim.y;
            const unsigned int loop_stride    = workgroup_size; // This stride should enable better memory access coalescing

            const unsigned int x_cluster_coordinate          = blockIdx.x;
            const unsigned int y_cluster_coordinate          = blockIdx.y;
            const unsigned int workgroup_dedicated_bin_index = x_cluster_coordinate * gridDim.y + y_cluster_coordinate; // The indexing order is: x * ywidth * zwidth + y * zwidth + z
            const unsigned int thread_index_offset           = threadIdx.x;

            // The unit is the cell
            const unsigned int global_x_scratch_space_coordinate_offset = x_cluster_coordinate * Params::getGPUClusterWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ );
            const unsigned int global_y_scratch_space_coordinate_offset = y_cluster_coordinate * Params::getGPUClusterWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ );

            // NOTE: We gain from the particles not being sorted inside a
            // cluster because it reduces the bank conflicts one gets when
            // multiple threads access the same part of the shared memory. Such
            // "conflicted" accesses are serialized !
            // NOTE: We use a bit to much LDS. For Jx, the first row could be
            // discarded, for Jy we could remove the first column.

            static constexpr unsigned int kFieldScratchSpaceSize = Params::getGPUInterpolationClusterCellVolume( 2 /* 2D */, 2 /* 2nd order interpolation */ );

            // NOTE: I tried having only one cache and reusing it. Doing that
            // requires you to iterate multiple time over the particle which is
            // possible but cost more bandwidth. The speedup was ~x0.92.
            __shared__ Float Jx_scratch_space[kFieldScratchSpaceSize];
            __shared__ Float Jy_scratch_space[kFieldScratchSpaceSize];
            __shared__ Float Jz_scratch_space[kFieldScratchSpaceSize];

            // Init the shared memory

            for( unsigned int field_index = thread_index_offset;
                 field_index < kFieldScratchSpaceSize;
                 field_index += workgroup_size ) {
                Jx_scratch_space[field_index] = 0.0;
                Jy_scratch_space[field_index] = 0.0;
                Jz_scratch_space[field_index] = 0.0;
            }

            __syncthreads();

            const unsigned int particle_count = device_bin_index[bin_count - 1];

            // This workgroup has to process distance(last_particle,
            // first_particle) particles
            const unsigned int first_particle = workgroup_dedicated_bin_index == 0 ? 0 :
                                                                                     device_bin_index[workgroup_dedicated_bin_index - 1];
            const unsigned int last_particle  = device_bin_index[workgroup_dedicated_bin_index];

            for( unsigned int particle_index = first_particle + thread_index_offset;
                 particle_index < last_particle;
                 particle_index += loop_stride ) {
                const Float invgf                         = static_cast<Float>( device_invgf_[particle_index] );
                const int *const __restrict__ iold        = &device_iold_[particle_index];
                const double *const __restrict__ deltaold = &device_deltaold_[particle_index];

                Float Sx0[5];
                Float Sx1[5];
                Float Sy0[5];
                Float Sy1[5];
                // double DSx[5];
                // double DSy[5];

                // Variable declaration & initialization
                // Esirkepov's paper: https://arxiv.org/pdf/physics/9901047.pdf

                // Locate the particle on the primal grid at former time-step & calculate coeff. S0
                {
                    const Float delta  = deltaold[0 * particle_count];
                    const Float delta2 = delta * delta;
                    Sx0[0]             = static_cast<Float>( 0.0 );
                    Sx0[1]             = static_cast<Float>( 0.5 ) * ( delta2 - delta + static_cast<Float>( 0.25 ) );
                    Sx0[2]             = static_cast<Float>( 0.75 ) - delta2;
                    Sx0[3]             = static_cast<Float>( 0.5 ) * ( delta2 + delta + static_cast<Float>( 0.25 ) );
                    Sx0[4]             = static_cast<Float>( 0.0 );
                }
                {
                    const Float delta  = deltaold[1 * particle_count];
                    const Float delta2 = delta * delta;
                    Sy0[0]             = static_cast<Float>( 0.0 );
                    Sy0[1]             = static_cast<Float>( 0.5 ) * ( delta2 - delta + static_cast<Float>( 0.25 ) );
                    Sy0[2]             = static_cast<Float>( 0.75 ) - delta2;
                    Sy0[3]             = static_cast<Float>( 0.5 ) * ( delta2 + delta + static_cast<Float>( 0.25 ) );
                    Sy0[4]             = static_cast<Float>( 0.0 );
                }

                // Locate the particle on the primal grid at current time-step & calculate coeff. S1
                {
                    const Float xpn = static_cast<Float>( device_particle_position_x[particle_index] ) * dx_inv;
                    const int   ip  = std::round( xpn );
                    // const int    ip       = static_cast<int>( xpn + 0.5 ); // std::round | rounding approximation which is correct enough and faster in this case
                    const int   ipo      = iold[0 * particle_count];
                    const int   ip_m_ipo = ip - ipo - i_domain_begin;
                    const Float delta    = xpn - static_cast<Float>( ip );
                    const Float delta2   = delta * delta;

                    Sx1[0] = static_cast<Float>( 0.0 );
                    Sx1[1] = static_cast<Float>( 0.0 );
                    // Sx1[2] = 0.0; // Always set below
                    Sx1[3] = static_cast<Float>( 0.0 );
                    Sx1[4] = static_cast<Float>( 0.0 );

                    Sx1[ip_m_ipo + 1] = static_cast<Float>( 0.5 ) * ( delta2 - delta + static_cast<Float>( 0.25 ) );
                    Sx1[ip_m_ipo + 2] = static_cast<Float>( 0.75 ) - delta2;
                    Sx1[ip_m_ipo + 3] = static_cast<Float>( 0.5 ) * ( delta2 + delta + static_cast<Float>( 0.25 ) );
                }
                {
                    const Float ypn = static_cast<Float>( device_particle_position_y[particle_index] ) * dy_inv;
                    const int   jp  = std::round( ypn );
                    // const int    jp       = static_cast<int>( ypn + 0.5 ); // std::round | rounding approximation which is correct enough and faster in this case
                    const int   jpo      = iold[1 * particle_count];
                    const int   jp_m_jpo = jp - jpo - j_domain_begin;
                    const Float delta    = ypn - static_cast<Float>( jp );
                    const Float delta2   = delta * delta;

                    Sy1[0] = static_cast<Float>( 0.0 );
                    Sy1[1] = static_cast<Float>( 0.0 );
                    // Sy1[2] = 0.0; // Always set below
                    Sy1[3] = static_cast<Float>( 0.0 );
                    Sy1[4] = static_cast<Float>( 0.0 );

                    Sy1[jp_m_jpo + 1] = static_cast<Float>( 0.5 ) * ( delta2 - delta + static_cast<Float>( 0.25 ) );
                    Sy1[jp_m_jpo + 2] = static_cast<Float>( 0.75 ) - delta2;
                    Sy1[jp_m_jpo + 3] = static_cast<Float>( 0.5 ) * ( delta2 + delta + static_cast<Float>( 0.25 ) );
                }

                // DSx[0] = Sx1[0] - Sx0[0];
                // DSx[1] = Sx1[1] - Sx0[1];
                // DSx[2] = Sx1[2] - Sx0[2];
                // DSx[3] = Sx1[3] - Sx0[3];
                // DSx[4] = Sx1[4] - Sx0[4];

                // DSy[0] = Sy1[0] - Sy0[0];
                // DSy[1] = Sy1[1] - Sy0[1];
                // DSy[2] = Sy1[2] - Sy0[2];
                // DSy[3] = Sy1[3] - Sy0[3];
                // DSy[4] = Sy1[4] - Sy0[4];

                // (x,y,z) components of the current density for the macro-particle
                const Float charge_weight = inv_cell_volume * static_cast<Float>( device_particle_charge[particle_index] ) * static_cast<Float>( device_particle_weight[particle_index] );
                const Float crx_p         = charge_weight * dx_ov_dt;
                const Float cry_p         = charge_weight * dy_ov_dt;
                const Float crz_p         = charge_weight * static_cast<Float>( 1.0 / 3.0 ) * static_cast<Float>( device_particle_momentum_z[particle_index] ) * invgf;

                // This is the particle position as grid index
                // This minus 2 come from the order 2 scheme, based on a 5 points stencil from -2 to +2.
                const int ipo = iold[0 * particle_count] -
                                2 /* Offset so we dont uses negative numbers in the loop */ -
                                global_x_scratch_space_coordinate_offset /* Offset to get cluster relative coordinates */;
                const int jpo = iold[1 * particle_count] -
                                2 /* Offset so we dont uses negative numbers in the loop */ -
                                global_y_scratch_space_coordinate_offset /* Offset to get cluster relative coordinates */;

                // // Atomic optimized loops
                // // Jx

                // for( unsigned int i = 1; i < 5; ++i ) {
                //     const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                //     for( unsigned int j = 0; j < 5; ++j ) {
                //         tmpJx[j] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( Sy0[j] + static_cast<Float>( 0.5 ) * ( Sy1[j] - Sy0[j] ) );
                //         ::atomicAdd( &Jx_scratch_space[iloc + j], tmpJx[j] );
                //     }
                // }

                // // Jy

                // for( unsigned int i = 0; i < 5; ++i ) {
                //     const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                //     Float     tmp{};
                //     for( unsigned int j = 1; j < 5; ++j ) {
                //         tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + static_cast<Float>( 0.5 ) * ( Sx1[i] - Sx0[i] ) );
                //         ::atomicAdd( &Jy_scratch_space[iloc + j], tmp );
                //     }
                // }

                // // Jz

                // for( unsigned int i = 0; i < 5; ++i ) {
                //     const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                //     for( unsigned int j = 0; j < 5; ++j ) {
                //         ::atomicAdd( &Jz_scratch_space[iloc + j], crz_p * ( Sy0[j] * ( static_cast<Float>( 0.5 ) * Sx1[i] + Sx0[i] ) +
                //                                                             Sy1[j] * ( static_cast<Float>( 0.5 ) * Sx0[i] + Sx1[i] ) ) );
                //     }
                // }

                // Atomic optimized loops
                // Jx

                Float tmpJx[5]{};

                for( unsigned int i = 1; i < 5; ++i ) {
                    const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    tmpJx[0] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( static_cast<Float>( 0.5 ) * ( Sy1[0] - Sy0[0] ) );
                    atomicAdd( &Jx_scratch_space[iloc], tmpJx[0] );
                    for( unsigned int j = 1; j < 5; ++j ) {
                        tmpJx[j] -= crx_p * ( Sx1[i - 1] - Sx0[i - 1] ) * ( Sy0[j] + static_cast<Float>( 0.5 ) * ( Sy1[j] - Sy0[j] ) );
                        atomicAdd( &Jx_scratch_space[iloc + j], tmpJx[j] );
                    }
                }

                // Jy

                for( unsigned int i = 0; i < 1; ++i ) {
                    const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    Float     tmp{};
                    for( unsigned int j = 1; j < 5; j++ ) {
                        tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + static_cast<Float>( 0.5 ) * ( Sx1[i] - Sx0[i] ) );
                        atomicAdd( &Jy_scratch_space[iloc + j], tmp );
                    }
                }

                for( unsigned int i = 1; i < 5; ++i ) {
                    const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    Float     tmp{};
                    for( unsigned int j = 1; j < 5; ++j ) {
                        tmp -= cry_p * ( Sy1[j - 1] - Sy0[j - 1] ) * ( Sx0[i] + static_cast<Float>( 0.5 ) * ( Sx1[i] - Sx0[i] ) );
                        atomicAdd( &Jy_scratch_space[iloc + j], tmp );
                    }
                }

                // Jz

                for( unsigned int i = 0; i < 1; ++i ) {
                    const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    atomicAdd( &Jz_scratch_space[iloc], crz_p * ( Sy1[0] * ( /* 0.5 * Sx0[i] + */ Sx1[i] ) ) );
                    for( unsigned int j = 1; j < 5; j++ ) {
                        atomicAdd( &Jz_scratch_space[iloc + j], crz_p * ( Sy0[j] * ( static_cast<Float>( 0.5 ) * Sx1[i] /* + Sx0[i] */ ) +
                                                                          Sy1[j] * ( /* 0.5 * Sx0[i] + */ Sx1[i] ) ) );
                    }
                }

                for( unsigned int i = 1; i < 5; ++i ) {
                    const int iloc = ( i + ipo ) * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + jpo;
                    atomicAdd( &Jz_scratch_space[iloc], crz_p * ( Sy1[0] * ( static_cast<Float>( 0.5 ) * Sx0[i] + Sx1[i] ) ) );
                    for( unsigned int j = 1; j < 5; ++j ) {
                        atomicAdd( &Jz_scratch_space[iloc + j], crz_p * ( Sy0[j] * ( static_cast<Float>( 0.5 ) * Sx1[i] + Sx0[i] ) +
                                                                          Sy1[j] * ( static_cast<Float>( 0.5 ) * Sx0[i] + Sx1[i] ) ) );
                    }
                }
            }

            __syncthreads();

            for( unsigned int field_index = thread_index_offset;
                 field_index < kFieldScratchSpaceSize;
                 field_index += workgroup_size ) {

                // The indexing order is: x * ywidth * zwidth + y * zwidth + z
                const unsigned int local_x_scratch_space_coordinate = field_index / Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ );
                const unsigned int local_y_scratch_space_coordinate = field_index % Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ );

                const unsigned int global_x_scratch_space_coordinate = global_x_scratch_space_coordinate_offset + local_x_scratch_space_coordinate;
                const unsigned int global_y_scratch_space_coordinate = global_y_scratch_space_coordinate_offset + local_y_scratch_space_coordinate;

                // The indexing order is: x * ywidth * zwidth + y * zwidth + z
                const unsigned int global_memory_index = global_x_scratch_space_coordinate * nprimy + global_y_scratch_space_coordinate;
                const unsigned int scratch_space_index = field_index; // local_x_scratch_space_coordinate * Params::getGPUClusterWithGhostCellWidth( 2 /* 2D */, 2 /* 2nd order interpolation */ ) + local_y_scratch_space_coordinate;

                // These atomics are basically free (very few of them).
                ::atomicAdd( &device_Jx[global_memory_index], static_cast<double>( Jx_scratch_space[scratch_space_index] ) );
                ::atomicAdd( &device_Jy[global_memory_index + /* We handle the FTDT/picsar */ pxr * global_x_scratch_space_coordinate], static_cast<double>( Jy_scratch_space[scratch_space_index] ) );
                ::atomicAdd( &device_Jz[global_memory_index], static_cast<double>( Jz_scratch_space[scratch_space_index] ) );
            }
        }
    } // namespace kernel

    static inline void
    currentDepositionKernel( double *__restrict__ host_Jx,
                             double *__restrict__ host_Jy,
                             double *__restrict__ host_Jz,
                             int Jx_size,
                             int Jy_size,
                             int Jz_size,
                             const double *__restrict__ device_particle_position_x,
                             const double *__restrict__ device_particle_position_y,
                             const double *__restrict__ device_particle_momentum_z,
                             const short *__restrict__ device_particle_charge,
                             const double *__restrict__ device_particle_weight,
                             const int *__restrict__ host_bin_index,
                             unsigned int x_dimension_bin_count,
                             unsigned int y_dimension_bin_count,
                             const double *__restrict__ host_invgf_,
                             const int *__restrict__ host_iold_,
                             const double *__restrict__ host_deltaold_,
                             double inv_cell_volume,
                             double dx_inv,
                             double dy_inv,
                             double dx_ov_dt,
                             double dy_ov_dt,
                             int    i_domain_begin,
                             int    j_domain_begin,
                             int    nprimy,
                             int    pxr )
    {
        int device_count;
        checkHIPErrors( ::hipGetDeviceCount( &device_count ) );
        SMILEI_ASSERT( device_count == 1 );

        // NOTE:
        // This cluster is very strongly bound by atomic operations in LDS (shared memory)
        // TODO(Etienne M): Find a way to lessen the atomic usage

        const ::dim3 kGridDimensionInBlock{ static_cast<uint32_t>( x_dimension_bin_count ), static_cast<uint32_t>( y_dimension_bin_count ), 1 };
        // On an MI100:
        // 448 for F32 and 4x4 cluster width | past 128, the block size does not matter, we are atomic bound anyway
        // 128 for F64 and 4x4 cluster width | atomic bound
        const ::dim3 kBlockDimensionInWorkItem{ 128, 1, 1 };

        using Float = double; // float/double

        auto KernelFunction = kernel::depositeForAllCurrentDimensions<Float>;

        hipLaunchKernelGGL( KernelFunction,
                            kGridDimensionInBlock,
                            kBlockDimensionInWorkItem,
                            0, // Shared memory
                            0, // Stream
                            // Kernel arguments
                            smilei::tools::gpu::HostDeviceMemoryManagment::GetDevicePointer( host_Jx ),
                            smilei::tools::gpu::HostDeviceMemoryManagment::GetDevicePointer( host_Jy ),
                            smilei::tools::gpu::HostDeviceMemoryManagment::GetDevicePointer( host_Jz ),
                            Jx_size, Jy_size, Jz_size,
                            device_particle_position_x,
                            device_particle_position_y,
                            device_particle_momentum_z,
                            device_particle_charge,
                            device_particle_weight,
                            smilei::tools::gpu::HostDeviceMemoryManagment::GetDevicePointer( host_bin_index ),
                            smilei::tools::gpu::HostDeviceMemoryManagment::GetDevicePointer( host_invgf_ ),
                            smilei::tools::gpu::HostDeviceMemoryManagment::GetDevicePointer( host_iold_ ),
                            smilei::tools::gpu::HostDeviceMemoryManagment::GetDevicePointer( host_deltaold_ ),
                            inv_cell_volume,
                            dx_inv, dy_inv,
                            dx_ov_dt, dy_ov_dt,
                            i_domain_begin, j_domain_begin,
                            nprimy,
                            pxr );

        checkHIPErrors( ::hipDeviceSynchronize() );
    }

} // namespace hip

    #endif

//! Project global current densities (EMfields->Jx_/Jy_/Jz_)
//!
extern "C" void
currentDepositionKernel( double *__restrict__ host_Jx,
                         double *__restrict__ host_Jy,
                         double *__restrict__ host_Jz,
                         int Jx_size,
                         int Jy_size,
                         int Jz_size,
                         const double *__restrict__ device_particle_position_x,
                         const double *__restrict__ device_particle_position_y,
                         const double *__restrict__ device_particle_momentum_z,
                         const short *__restrict__ device_particle_charge,
                         const double *__restrict__ device_particle_weight,
                         const int *__restrict__ host_bin_index,
                         unsigned int x_dimension_bin_count,
                         unsigned int y_dimension_bin_count,
                         const double *__restrict__ host_invgf_,
                         const int *__restrict__ host_iold_,
                         const double *__restrict__ host_deltaold_,
                         double inv_cell_volume,
                         double dx_inv,
                         double dy_inv,
                         double dx_ov_dt,
                         double dy_ov_dt,
                         int    i_domain_begin,
                         int    j_domain_begin,
                         int    nprimy,
                         int    pxr )
{
    #if defined( PRIVATE_SMILEI_USE_OPENMP_PROJECTION_IMPLENTATION )
    naive:: // the naive, OMP version serves as a referance along with the CPU version
    #else
    hip::
    #endif
        currentDepositionKernel( host_Jx, host_Jy, host_Jz,
                                 Jx_size, Jy_size, Jz_size,
                                 device_particle_position_x, device_particle_position_y,
                                 device_particle_momentum_z,
                                 device_particle_charge,
                                 device_particle_weight,
                                 host_bin_index,
                                 x_dimension_bin_count,
                                 y_dimension_bin_count,
                                 host_invgf_,
                                 host_iold_, host_deltaold_,
                                 inv_cell_volume,
                                 dx_inv, dy_inv,
                                 dx_ov_dt, dy_ov_dt,
                                 i_domain_begin, j_domain_begin,
                                 nprimy,
                                 pxr );
}

#endif
